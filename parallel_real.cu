#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <omp.h>
#include <malloc.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
typedef struct {
    int x;
}elem;
__device__ int MFKnapsack(int, int); //Function returning the optimal value for a given n and W
int max(int, int);
int n;
int* weights;
int* d_weights;
int* values;
int* d_values;
int W;
elem **F;
elem **d_F;
int main()
{
    clock_t start, end;

    printf("enter number of Questions in the Exam paper :");
	scanf("%d", &n);
	printf("---------------------------------------------------");
	//Allocating host memory
	weights = (int*)malloc((n+1)*sizeof(int));
	values = (int*)malloc((n+1)*sizeof(int));
	
	//Allocating device memory
	hipMalloc((void**)&d_weights, sizeof(int)*(n+1));
	hipMalloc((void**)&d_values, sizeof(int)*(n+1));
	
	int i, j;
	//Read weights and corresponding values for 'n' items
	printf("\nenter marks (maximum) for all %d questions\n \n",n);
	for(i=1;i<=n;i++)
	{
	    printf("enter marks (maximum) for question number %d :",i);
		scanf("%d", &weights[i]);
	}
	printf("---------------------------------------------------");
	printf("\nenter marks scored by the student for all %d questions \n",n);
	for(i=1;i<=n;i++)
	{
	    printf("enter marks scored by student in question number %d :",i);
		scanf("%d", &values[i]);
	}
    printf("---------------------------------------------------");
	//Read Max. Weight Capacity of Knapsack
	printf("\nenter the total(maximum) marks of th paper : ");
	scanf("%d", &W);
	//Allocating memory for the Memory Function in the host memory 
	F = (elem**)malloc((n+1)*sizeof(elem*));
	
	//Allocating memory for the Memory Function in the host memory 
	hipMalloc((elem**)&d_F, sizeof(elem*)*(n+1));
	
	
	//64 => boundary width to the next element
	for(i=0;i<=n;i++)
		F[i] = (elem*)_aligned_malloc((W+1)*sizeof(elem),64);
		
		
	//Zeroing out the first row
	for(i=0;i<(W+1);i++)
		(*(F) + i ) -> x = 0;
	//Zeroing out the first column
	for(i=0;i<(n+1);i++)
		(*(F+i)) -> x = 0;
	for(i=1;i<=n;i++)
	{
		for(j=1;j<=W;j++)
			(*(F + i) + j) -> x = -1;
	}
    start = clock();
    hipMemcpy(d_weights, weights, sizeof(int)*(n+1), hipMemcpyHostToDevice);
    hipMemcpy(d_values, values , sizeof(int)*(n+1), hipMemcpyHostToDevice);
    hipMemcpy(d_F, F , sizeof(int)*(n+1), hipMemcpyHostToDevice);
	int res;
	res=MFKnapsack<<<1,256>>>(n, W); //Optimal value for items with given weights and values
	end = clock();
    
	hipMemcpy(weights,d_weights, sizeof(int)*(n+1), hipMemcpyDeviceToHost);
    hipMemcpy( values ,d_values, sizeof(int)*(n+1), hipMemcpyDeviceToHost);
    hipMemcpy( F ,d_F, sizeof(int)*(n+1), hipMemcpyDeviceToHost);
    
	double time_taken = (double)(end - start) / (double)(CLOCKS_PER_SEC);
	//printf("Total marks of the student= %d\n", res);
	printf("Time taken = %1f s",(double)(time_taken));
	
	//De allocating device memory
	hipFree(d_weights);
	hipFree(d_values);

	
	//De allocating host memory
	free(weights);
	free(values);
	return 0;
}
__device__ int MFKnapsack(int i, int j)
{
	int value;
	if( ((*(F + i) + j)->x) < 0)
	{
		if(j < d_weights[i])  //If weight of item is more than current capacity
			value = MFKnapsack<<<1,256>>>(i-1, j); //Value of previous item
		else
		{
			int a;
			int b;
			a = MFKnapsack<<<1,256>>>(i-1, j);		
			b =  (d_values[i] + MFKnapsack<<<1,256>>>(i-1, j - d_weights[i]));			
			value = max(a,b);
		}
		(*(F + i) + j)->x = value;
	}
	//printf("\nTotal marks of the student= %d \n",();
	return (*(F + i) + j)->x);
}
int max(int a, int b)
{
	return (a>b?a:b);
}

