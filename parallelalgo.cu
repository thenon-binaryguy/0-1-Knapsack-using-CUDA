#include<stdio.h>
#include<omp.h>
#include <time.h>
#include <malloc.h>
#include <hip/hip_runtime.h>




__global__ void Knapsack(int W, int wt[], int val[], int n,int *res)
{
    
	int i, w;
    int** K = new int*[n+1];
    //int K[n + 1][W + 1];
    for(int i = 0; i < (n+1); i++)
    {K[i] = new int[W+1];}


    for (i = 0; i <= n; i++)
    {
        for (w = 0; w <= W; w++)
        {
            if (i == 0 || w == 0)
                K[i][w] = 0;
            else if (wt[i - 1] <= w)
                K[i][w] = max(val[i - 1]
                          + K[i - 1][w - wt[i - 1]],
                          K[i - 1][w]);
            else
                K[i][w] = K[i - 1][w];
        }
    }
 	//printf("Total marks of the student= %d\n", K[n][W]);
 	int q;
 	q=K[n][W];
 	*res=q;
 	delete[] K ;
}	
int main(void)
{
    int w,n;
	int i;
    int* weights;
	int* values;
	int* d_values;
	int* d_weights;
	int *res,*r;
	clock_t start, end;
    printf("Parallel Execution Using Cuda \n-------------------------------------------------------");
    
    printf("\nEnter no. of questions, n = ");
    scanf("%d",&n);
    
    //Allocating host memory
    weights = (int*)malloc((n+1)*sizeof(int));
	values = (int*)malloc((n+1)*sizeof(int));
	r=(int*)malloc(sizeof(int));
	
	//Allocating device memory
	hipMalloc((void**)&d_weights, sizeof(int)*(n+1));
	hipMalloc((void**)&d_values, sizeof(int)*(n+1));
	hipMalloc((void**)&res, sizeof(int));
	
    printf("\nEnter maximum marks for all questions (weight array) : \n");
    for(i=0;i<n;i++)
    {
        printf("Maximum marks for question #%d = ",i);
        scanf("%d",&weights[i]);
    }
    printf("\nEnter marks scored by student (value array) : \n");
    for(i=0;i<n;i++)
    {
        printf("Marks scored by student in question #%d = ",i);
        scanf("%d",&values[i]);
    }
    printf("\nEnter total(maximum) marks of the paper(Weight w), w = ");
    scanf("%d",&w);
    //double beg = omp_get_wtime();
    
    hipMemcpy(d_weights, weights, sizeof(int)*(n+1), hipMemcpyHostToDevice);
    hipMemcpy(d_values, values , sizeof(int)*(n+1), hipMemcpyHostToDevice);
    
    start = clock();
    //Executing the Kernel
	Knapsack<<<1,256>>>(w,d_weights,d_values,n,res); //Optimal valuesue for items with given weights and values
	
    hipMemcpy(r,res,sizeof(int),hipMemcpyDeviceToHost);
	printf("Total marks of the student= %d\n", *r);
	end = clock();
    double time_taken = (double)(end - start) / (double)(CLOCKS_PER_SEC);
	printf("Time taken = %1f s",(double)(time_taken));
	
	hipMemcpy(weights,d_weights, sizeof(int)*(n+1), hipMemcpyDeviceToHost);
    hipMemcpy( values ,d_values, sizeof(int)*(n+1),hipMemcpyDeviceToHost);
    
	//De allocating device memory
	hipFree(d_weights);
	hipFree(d_values);

	
	//De allocating host memory
	free(weights);
	free(values);
	
	return 0;
}
